#include "hip/hip_runtime.h"
/**
 * Main module of the CUES project.
 */

#include <cstdio>
#include <cstdlib>
#include <hiprand.h>

#include "utils.cuh"
#include "graph.cuh"
#include "simulation.cuh"

__device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
/*int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}*/

int main(int argc, char **argv) {
	if(argc != 6) {
		printUsage();
		exit(1);
	}

	char *graphPath = argv[1];
	int patientZero = atoi(argv[2]);
	double p = atof(argv[3]);
	double q = atof(argv[4]);
	int simulations = atoi(argv[5]);

	exitIf(p < 0.0 || p > 1.0,
			"The p parameter should be from the interval [0-1].");
	exitIf(q < 0.0 || q > 1.0,
			"The q parameter should be from the interval [0-1].");
	exitIf(simulations < 1, "The number of simulations should be at least 1.");

	printf("Input parameters:\n");

	printf("\t%-25s %s\n", "Graph file:", graphPath);
	printf("\t%-25s %d\n", "Source node:", patientZero);
	printf("\t%-25s %.2f\n", "Q:", q);
	printf("\t%-25s %.2f\n", "P:", p);
	printf("\t%-25s %d\n", "Simulations:", simulations);

	printf("\nSearching for best device... ");

	int devideID = cudaGetMaxGflopsDeviceID();
	CUDA_CHECK_RETURN(hipSetDevice(devideID));

	hipDeviceProp_t deviceProperties;
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProperties, devideID));

	int deviceMajor = deviceProperties.major;
	int deviceMinor = deviceProperties.minor;
	int deviceMPs = deviceProperties.multiProcessorCount;
	int residentThreadsPerMP = deviceProperties.maxThreadsPerMultiProcessor;
	int totalResidentThreads = deviceMPs * residentThreadsPerMP;

	printf("DONE\n");
	printf("\t%-25s %s\n", "Device:", deviceProperties.name);
	printf("\t%-25s %d.%d\n", "Capability", deviceMajor, deviceMinor);
	printf("\t%-25s %d\n", "Multiprocessors", deviceMPs);
	printf("\t%-25s %d\n", "Total CUDA cores", deviceMPs
			* convertSMVersion2Cores(deviceMajor, deviceMinor));
	printf("\t%-25s %d\n", "Total resident threads:", totalResidentThreads);

	printf("\nLoading graph... ");
	Graph *graph = loadGraph(graphPath);
	printf("DONE\n");

	exitIf(patientZero < 0 || patientZero > graph->N - 1,
			"Source node is not present in the input graph.");

	printf("\t%-25s %u\n", "Nodes:", graph->N);
	printf("\t%-25s %u\n", "Edges:", graph->M);

	printf("\nCreating simulation context... ");
	SimulationContext *context = createSimulationContext(graph);
	printf("DONE\n\n");

	for(int simulation = 1; simulation <= simulations; simulation++) {
		printf("Running %d. simulation... ", simulation);

		prepareSimulationContext(context, patientZero);

		// DO STUFF

		printf(" DONE\n");
	}

	freeSimulationContext(context);
	freeGraph(graph);
}
