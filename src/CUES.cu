#include "hip/hip_runtime.h"
/**
 * Main module of the CUES project.
 */

#include <cstdio>
#include <cstdlib>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.cuh"
#include "graph.cuh"
#include "simulation.cuh"
#include "kernels.cuh"

__device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
/*int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}*/

int main(int argc, char **argv) {
	if(argc != 6) {
		printUsage();
		exit(1);
	}

	char *graphPath = argv[1];
	int patientZero = atoi(argv[2]);
	double p = atof(argv[3]);
	double q = atof(argv[4]);
	int simulations = atoi(argv[5]);

	exitIf(p < 0.0 || p > 1.0,
			"The p parameter should be from the interval [0-1].");
	exitIf(q < 0.0 || q > 1.0,
			"The q parameter should be from the interval [0-1].");
	exitIf(simulations < 1, "The number of simulations should be at least 1.");

	printf("Input parameters:\n");

	printf("\t%-25s %s\n", "Graph file:", graphPath);
	printf("\t%-25s %d\n", "Source node:", patientZero);
	printf("\t%-25s %.2f\n", "Q:", q);
	printf("\t%-25s %.2f\n", "P:", p);
	printf("\t%-25s %d\n", "Simulations:", simulations);

	printf("\nSearching for best device... ");

	int devideID = cudaGetMaxGflopsDeviceID();
	CUDA_CHECK_RETURN(hipSetDevice(devideID));

	hipDeviceProp_t deviceProperties;
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProperties, devideID));

	int deviceMajor = deviceProperties.major;
	int deviceMinor = deviceProperties.minor;
	int deviceMPs = deviceProperties.multiProcessorCount;
	int residentThreadsPerMP = deviceProperties.maxThreadsPerMultiProcessor;
	int totalResidentThreads = deviceMPs * residentThreadsPerMP;

	printf("DONE\n");
	printf("\t%-25s %s\n", "Device:", deviceProperties.name);
	printf("\t%-25s %d.%d\n", "Capability", deviceMajor, deviceMinor);
	printf("\t%-25s %d\n", "Multiprocessors", deviceMPs);
	printf("\t%-25s %d\n", "Total CUDA cores", deviceMPs
			* convertSMVersion2Cores(deviceMajor, deviceMinor));
	printf("\t%-25s %d\n", "Total resident threads:", totalResidentThreads);

	printf("\nLoading graph... ");
	Graph *graph = loadGraph(graphPath);
	printf("DONE\n");

	printIntArray((int *)graph->R, graph->RSize, true);
	printIntArray((int *) graph->C, graph->CSize, true);

	exitIf(patientZero < 0 || patientZero > graph->N - 1,
			"Source node is not present in the input graph.");

	printf("\t%-25s %u\n", "Nodes:", graph->N);
	printf("\t%-25s %u\n", "Edges:", graph->M);

	printf("\nCreating simulation context... ");
	SimulationContext *context = createSimulationContext(graph);
	printf("DONE\n");

	printf("\nInitializing random generator states... ");
	initRandoms<<<MAX_GRID_SIZE, BLOCK_SIZE>>>(context->randStates, SEED);
	printf("DONE\n\n");

	for(int simulation = 1; simulation <= simulations; simulation++) {
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		printf("Running %d. simulation... ", simulation);
		prepareSimulationContext(context, patientZero);

		int iteration = 0;
		unsigned int inputSize = 1;

		do {
			unsigned int blocks = (inputSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
			blocks = min(blocks, MAX_GRID_SIZE);

			printf("Input size, blocks: %d %d\n", inputSize, blocks);

			generateRandForFrontier<<<blocks, BLOCK_SIZE>>>(
					context->randStates,
					context->inputFrontier,
					context->inFrontierSize,
					context->pRand,
					context->qRand
			);

			contractExpand<<<blocks, BLOCK_SIZE>>>(
					iteration,
					p,
					q,
					context->nodes,
					context->R,
					context->C,
					context->inFrontierSize,
					context->inputFrontier,
					context->outFrontierSize,
					context->outputFrontier,
					context->infected,
					context->immune,
					context->didInfectNeighbors,
					context->pRand,
					context->qRand
			);

			iterationDone(context);
			iteration++;

			inputSize = getInputFrontierSize(context);
			printf("Frontier size: %u -> ", inputSize);
			printIntArray((int *) context->inputFrontier, inputSize, false);
		} while(inputSize > 0); // inputSize != 0

		printIntArray(context->infected, context->nodes - 1, false);
		printBoolArray(context->immune, context->nodes - 1, false);
		printBoolArray(context->didInfectNeighbors, context->nodes - 1, false);
		printf(" DONE\n");
	}

	freeSimulationContext(context);
	freeGraph(graph);

	CUDA_CHECK_RETURN(hipDeviceReset());
}
